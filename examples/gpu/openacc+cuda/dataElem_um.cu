
#include <hip/hip_runtime.h>
#include <string.h>
#include <stdio.h>

struct DataElement
{
  char *name;
  int value;
  float foo;
};

__global__
void Kernel(DataElement *elem) {
  printf("CUDA On device: name=%s, value=%d\n", elem->name, elem->value);

  elem->name[0] = 'd';
  elem->value++;
  elem->foo = elem->foo * elem->foo;
}

void launch(DataElement *elem) {
  Kernel<<< 1, 1 >>>(elem);
  hipDeviceSynchronize();
}

void do_cuda(void)
{
  DataElement *e;
  hipMallocManaged((void**)&e, sizeof(DataElement));

  e->value = 10;
  e->foo = 42.0;
  hipMallocManaged((void**)&(e->name), sizeof(char) * (strlen("hello") + 1) );
  strcpy(e->name, "hello");

  launch(e);

  printf("CUDA On host: name=%s, value=%d, foo=%f\n", e->name, e->value, e->foo);

  hipFree(e->name);
  hipFree(e);
  return;
}
