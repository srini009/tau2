
#include <hip/hip_runtime.h>
#include <string.h>
#include <stdio.h>
#include <iostream>

struct DataElement
{
  char *name;
  int value;
};

__global__
void Kernel(DataElement *elem) {
  printf("On device: name=%s, value=%d\n", elem->name, elem->value);

  elem->name[0] = 'd';
  elem->value++;
}

void launch(DataElement *elem, hipStream_t &stream) {
  Kernel<<< 1, 1, 0, stream >>>(elem);
  //cudaDeviceSynchronize();
}

void iteration(hipStream_t &stream)
{
  DataElement *e;
  hipMallocManaged((void**)&e, sizeof(DataElement));

  e->value = 10;
  hipMallocManaged((void**)&(e->name), sizeof(char) * (strlen("hello") + 1) );
  strcpy(e->name, "hello");

  launch(e, stream);

  printf("On host: name=%s, value=%d\n", e->name, e->value);

  hipFree(e->name);
  hipFree(e);

}

int main(void)
{
	hipError_t err;
	int count = 0;
	err = hipGetDeviceCount(&count);
	std::cout << count << " devices found." << std::endl;

	for (int d=0;d<count;d++) {
		err = hipSetDevice(d);
		if (err != hipSuccess) {
			std::cout << "error setting device, #=" << hipGetErrorString(err) << std::endl;
		}
		hipDeviceProp_t deviceProp;
		err = hipGetDeviceProperties(&deviceProp, d);
		if (err != hipSuccess) {
			std::cout << "error getting device properties, #=" << hipGetErrorString(err) << std::endl;
		}
		std::cout << "Using device " << d << ", name: " << deviceProp.name << std::endl;
        for (int s = 0 ; s < 10 ; s++) {
	        hipStream_t stream;
		    err = hipStreamCreate(&stream);
	        if (err != hipSuccess) {
		        std::cout << "error in stream creation, #=" << hipGetErrorString(err) << std::endl;
	        }
            iteration(stream);
		    hipStreamDestroy(stream);
        }
    }
}
